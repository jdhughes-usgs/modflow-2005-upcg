
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

// number of threads per block
//const int numThreadsPerBlock = 256;
const int numThreadsPerBlock = 1024;

//// device to use in case there is more than one
//static int selectedDevice = 0;

__global__ void kern_Dvxv(const int n, double* v1, double* v2, double* v3)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < n)
	{
		v3[i] = v1[i] * v2[i];
		i += blockDim.x * gridDim.x;
	}
}


extern "C" {

	//element-wise vector multiplication double precision
	void cuda_Dvxv(const int* n, double* v1, double* v2, double* v3)
	{
		int numBlocks = (*n+(numThreadsPerBlock-1)) / numThreadsPerBlock;

		kern_Dvxv<<<numBlocks,numThreadsPerBlock>>>(*n,v1,v2,v3);

		return;
	}

}

